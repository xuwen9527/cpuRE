#include "hip/hip_runtime.h"
#include "bitmask.cuh"

namespace cpuRE {
  __device__
  int BitMask::count() {
    return __popcll(mask);
  }

  __device__
  glm::ivec2 BitMask::coord(int index) {
    index += 1;
    uint32_t p = 32;
    for (uint32_t offset = p / 2; offset > 0; offset /= 2)
      p = (__popcll(mask >> p) < index) ? (p - offset) : (p + offset);
    p = (__popcll(mask >> p) == index) ? p : p - 1;

    return { Cols - 1U - (p & (Cols - 1)), p >> ColsBits};
  }
}