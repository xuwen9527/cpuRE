#include "hip/hip_runtime.h"
#ifndef __TRIANGLE_BUFFER_H__
#define __TRIANGLE_BUFFER_H__

#include <glm/fwd.hpp>

namespace cpuRE {
  __device__
  void TriangleBuffer::allocate() {
    return atomicInc(&next_, NUM - 1);
  }
}

#endif